
#include <hip/hip_runtime.h>

// ParallelSPSS.Form1
extern "C" __global__  void addVector( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N);
// ParallelSPSS.Form1
extern "C" __global__  void compare_max( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N);
// ParallelSPSS.Form1
extern "C" __global__  void compare_min( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N);
// ParallelSPSS.Form1
extern "C" __global__  void sum( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N);

// ParallelSPSS.Form1
extern "C" __global__  void addVector( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += gridDim.x)
	{
		c[(i)] = a[(i)] + b[(i)];
	}
}
// ParallelSPSS.Form1
extern "C" __global__  void compare_max( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += gridDim.x)
	{
		if (a[(i)] > b[(i)])
		{
			c[(i)] = a[(i)];
		}
		else
		{
			c[(i)] = b[(i)];
		}
	}
}
// ParallelSPSS.Form1
extern "C" __global__  void compare_min( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += gridDim.x)
	{
		if (a[(i)] < b[(i)])
		{
			c[(i)] = a[(i)];
		}
		else
		{
			c[(i)] = b[(i)];
		}
	}
}
// ParallelSPSS.Form1
extern "C" __global__  void sum( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += gridDim.x)
	{
		c[(i)] = a[(i)] + b[(i)];
	}
}
